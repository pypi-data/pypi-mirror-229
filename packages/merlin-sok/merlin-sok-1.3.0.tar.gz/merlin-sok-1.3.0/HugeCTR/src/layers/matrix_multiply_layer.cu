/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <functional>
#include <layers/matrix_multiply_layer.hpp>
#include <utils.cuh>
#include <utils.hpp>

namespace HugeCTR {
template <typename T>
MatrixMultiplyLayer<T>::MatrixMultiplyLayer(const std::vector<core23::Tensor>& input_tensors,
                                            core23::Tensor& output_tensor,
                                            const std::shared_ptr<GPUResource>& gpu_resource)
    : Layer({}, {}, gpu_resource) {
  try {
    num_ = input_tensors.size();

    // error input checking
    dims_lhs_ = input_tensors[0].shape().dims();
    dims_rhs_ = input_tensors[1].shape().dims();
    const auto& dim_lhs = input_tensors[0].shape();
    const auto& dim_rhs = input_tensors[1].shape();

    if (num_ < 2) {
      HCTR_OWN_THROW(Error_t::WrongInput, "MatrixMultiplyLayer needs at least 2 input tensors");
    }
    if (dims_lhs_ < 2 || dims_rhs_ < 2) {
      HCTR_OWN_THROW(Error_t::WrongInput, "MatrixMultiplyLayer inputs should have at least 2 dims");
    }
    if (dims_lhs_ == 2 && dims_rhs_ == 3) {
      HCTR_CHECK_HINT(dim_rhs[0] == dim_lhs[1], "MatrixMultiplyLayer 2Dx3D invalid shape");
    } else if (dims_lhs_ != dims_rhs_) {
      HCTR_OWN_THROW(Error_t::WrongInput, "MatrixMultiplyLayer invalid input shape");
    }

    if (dim_rhs[dims_lhs_ - 2] != dim_lhs[dims_lhs_ - 1]) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "The last two dimension of the input tensors should be m x n, n x k");
    }

    for (size_t i = 0; i < num_; i++) {
      input_tensors_.push_back(input_tensors[i]);
    }

    int64_t m = input_tensors[0].shape().size(dims_lhs_ - 2);
    int64_t k = input_tensors[1].shape().size(dims_lhs_ - 1);
    if (dims_lhs_ == 2 && dims_rhs_ == 3) {
      k = dim_rhs[1] * dim_rhs[2];
    }
    core23::TensorParams out_params = input_tensors[0].my_params();

    if (dims_lhs_ == 2) {
      if (dims_rhs_ == 2) {
        std::vector<int64_t> out_shape = {m, k};
        output_tensor = core23::Tensor(out_params.shape(out_shape));
      } else {
        std::vector<int64_t> out_shape = {m, dim_rhs[1], dim_rhs[2]};
        output_tensor = core23::Tensor(out_params.shape(out_shape));
      }
    } else if (dims_lhs_ == 3) {
      if (dim_lhs[0] != dim_rhs[0]) {
        HCTR_OWN_THROW(Error_t::WrongInput, "3D input tensors must have the same batch size");
      }
      int64_t b = dim_lhs[0];
      std::vector<int64_t> out_shape = {b, m, k};
      output_tensor = core23::Tensor(out_params.shape(out_shape));
    } else if (dims_lhs_ == 4) {
      if (dim_lhs[0] != dim_rhs[0]) {
        HCTR_OWN_THROW(Error_t::WrongInput, "4D input tensors must have the same batch size");
      }
      if (dim_lhs[1] != dim_rhs[1]) {
        HCTR_OWN_THROW(Error_t::WrongInput, "4D input tensors must have the same second dim");
      }
      int64_t b = dim_lhs[0];
      int64_t num_head = dim_lhs[1];
      std::vector<int64_t> out_shape = {b, num_head, m, k};
      output_tensor = core23::Tensor(out_params.shape(out_shape));
    }
    output_tensors_.push_back(output_tensor);

    fprop_inputA_tensor23_ = core23::Tensor(out_params);

  } catch (const std::runtime_error& rt_err) {
    HCTR_LOG_S(ERROR, WORLD) << rt_err.what() << std::endl;
    throw;
  }
}

/*
  1. 2D: (m, n), (n, k)  and output: (m, k)
  2. 3D: (batch_size, m, n), (batch_size, n, k) and output will: (batch_size, m, k)
  3. 2D x 3D: (m, n) , (n, g, h), and output will be:  (m, g, h)
*/
template <typename T>
MatrixMultiplyLayer<T>::MatrixMultiplyLayer(
    const Tensors2<T>& in_tensors, Tensor2<T>& out_tensor,
    const std::shared_ptr<GeneralBuffer2<CudaAllocator>>& blobs_buff,
    const std::shared_ptr<GPUResource>& gpu_resource)
    : Layer(gpu_resource) {
  try {
    num_ = in_tensors.size();

    // error input checking
    dims_lhs_ = in_tensors[0].get_dimensions().size();
    dims_rhs_ = in_tensors[1].get_dimensions().size();
    const auto& dim_lhs = in_tensors[0].get_dimensions();
    const auto& dim_rhs = in_tensors[1].get_dimensions();

    if (num_ < 2) {
      HCTR_OWN_THROW(Error_t::WrongInput, "MatrixMultiplyLayer needs at least 2 input tensors");
    }
    if (dims_lhs_ < 2 || dims_rhs_ < 2) {
      HCTR_OWN_THROW(Error_t::WrongInput, "MatrixMultiplyLayer inputs should have at least 2 dims");
    }

    if (dims_lhs_ == 2 && dims_rhs_ == 3) {
      HCTR_CHECK_HINT(dim_rhs[0] == dim_lhs[1], "MatrixMultiplyLayer 2Dx3D invalid shape");
    } else if (dims_lhs_ != dims_rhs_) {
      HCTR_OWN_THROW(Error_t::WrongInput, "MatrixMultiplyLayer invalid input shape");
    }

    if (dim_rhs[dims_lhs_ - 2] != dim_lhs[dims_lhs_ - 1]) {
      HCTR_OWN_THROW(Error_t::WrongInput,
                     "The last two dimension of the input tensors should be m x n, n x k");
    }

    for (size_t i = 0; i < num_; i++) {
      in_tensors_.push_back(in_tensors[i]);
    }

    size_t m = dim_lhs[dims_lhs_ - 2];
    size_t k = dim_rhs[dims_rhs_ - 1];
    if (dims_lhs_ == 2 && dims_rhs_ == 3) {
      k = dim_rhs[1] * dim_rhs[2];
    }
    if (dims_lhs_ == 2) {
      if (dims_rhs_ == 2) {
        std::vector<size_t> out_dim = {m, k};
        blobs_buff->reserve(out_dim, &out_tensor);
      } else {
        std::vector<size_t> out_dim = {m, dim_rhs[1], dim_rhs[2]};
        blobs_buff->reserve(out_dim, &out_tensor);
      }
    } else if (dims_lhs_ == 3) {
      if (dim_lhs[0] != dim_rhs[0]) {
        HCTR_OWN_THROW(Error_t::WrongInput, "3D input tensors must have the same batch size");
      }
      size_t b = dim_lhs[0];
      std::vector<size_t> out_dim = {b, m, k};
      blobs_buff->reserve(out_dim, &out_tensor);
    } else if (dims_lhs_ == 4) {
      if (dim_lhs[0] != dim_rhs[0]) {
        HCTR_OWN_THROW(Error_t::WrongInput, "4D input tensors must have the same batch size");
      }
      if (dim_lhs[1] != dim_rhs[1]) {
        HCTR_OWN_THROW(Error_t::WrongInput, "4D input tensors must have the same second dim");
      }
      size_t b = dim_lhs[0];
      size_t num_head = dim_lhs[1];
      std::vector<size_t> out_dim = {b, num_head, m, k};
      blobs_buff->reserve(out_dim, &out_tensor);
    }

    out_tensors_.push_back(out_tensor);

    blobs_buff->reserve(dim_lhs, &fprop_inputA_);

  } catch (const std::runtime_error& rt_err) {
    HCTR_LOG_S(ERROR, WORLD) << rt_err.what() << std::endl;
    throw;
  }
}

template <typename T>
void MatrixMultiplyLayer<T>::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());

  if (input_tensors_.empty()) {
    T* in1 = in_tensors_[0].get_ptr();
    T* in2 = in_tensors_[1].get_ptr();
    T* out = out_tensors_[0].get_ptr();

    const auto& in_tensor_dim = in_tensors_[0].get_dimensions();
    const auto& out_tensor_dim = out_tensors_[0].get_dimensions();

    size_t m, n, k, b = 1;

    b = dims_lhs_ == 3 ? in_tensor_dim[0] : 1;
    b = dims_lhs_ == 4 ? in_tensor_dim[0] * in_tensor_dim[1] : b;
    m = in_tensor_dim[dims_lhs_ - 2];
    n = in_tensor_dim[dims_lhs_ - 1];
    k = out_tensor_dim[dims_lhs_ - 1];
    if (dims_lhs_ == 2 && dims_rhs_ == 3) {
      k = out_tensor_dim[1] * out_tensor_dim[2];
    }
    float alpha = 1.0f, beta = 0.0f;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;

    for (size_t i = 0; i < b; i++) {
      T* cur_in1 = in1 + i * m * n;
      T* cur_in2 = in2 + i * n * k;
      T* cur_out = out + i * m * k;
      HCTR_LIB_THROW(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N, k, m, n,
                                  &alpha, cur_in2, HIP_R_32F, k, cur_in1, HIP_R_32F, n, &beta,
                                  cur_out, HIP_R_32F, k, compute_type, HIPBLAS_GEMM_DEFAULT));
    }

    HCTR_LIB_THROW(hipMemcpyAsync((void*)fprop_inputA_.get_ptr(), (void*)in1,
                                   in_tensors_[0].get_size_in_bytes(), hipMemcpyDeviceToDevice,
                                   get_gpu().get_stream()));
  } else {
    T* in1 = input_tensors_[0].data<T>();
    T* in2 = input_tensors_[1].data<T>();
    T* out = output_tensors_[0].data<T>();

    const auto& input_tensor_shape = input_tensors_[0].shape();
    const auto& output_tensor_shape = output_tensors_[0].shape();

    int64_t m, n, k, b = 1;

    b = dims_lhs_ == 3 ? input_tensor_shape.size(0) : 1;
    b = dims_lhs_ == 4 ? input_tensor_shape.size(0) * input_tensor_shape.size(1) : b;
    m = input_tensor_shape.size(dims_lhs_ - 2);
    n = input_tensor_shape.size(dims_lhs_ - 1);
    k = output_tensor_shape.size(dims_lhs_ - 1);
    if (dims_lhs_ == 2 && dims_rhs_ == 3) {
      k = output_tensor_shape[1] * output_tensor_shape[2];
    }
    float alpha = 1.0f, beta = 0.0f;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;

    for (auto i = 0; i < b; i++) {
      T* cur_in1 = in1 + i * m * n;
      T* cur_in2 = in2 + i * n * k;
      T* cur_out = out + i * m * k;
      HCTR_LIB_THROW(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N, k, m, n,
                                  &alpha, cur_in2, HIP_R_32F, k, cur_in1, HIP_R_32F, n, &beta,
                                  cur_out, HIP_R_32F, k, compute_type, HIPBLAS_GEMM_DEFAULT));
    }

    HCTR_LIB_THROW(hipMemcpyAsync(fprop_inputA_tensor23_.data(), (void*)in1,
                                   input_tensors_[0].num_bytes(), hipMemcpyDeviceToDevice,
                                   get_gpu().get_stream()));
  }
}

template <typename T>
void MatrixMultiplyLayer<T>::bprop() {
  CudaDeviceContext context(get_device_id());

  if (input_tensors_.empty()) {
    T* in1 = in_tensors_[0].get_ptr();
    T* in2 = in_tensors_[1].get_ptr();
    T* out = out_tensors_[0].get_ptr();

    const auto& in_tensor_dim = in_tensors_[0].get_dimensions();
    const auto& out_tensor_dim = out_tensors_[0].get_dimensions();

    size_t m, n, k, b = 1;

    b = dims_lhs_ == 3 ? in_tensor_dim[0] : 1;
    b = dims_lhs_ == 4 ? in_tensor_dim[0] * in_tensor_dim[1] : b;
    m = in_tensor_dim[dims_lhs_ - 2];
    n = in_tensor_dim[dims_lhs_ - 1];
    k = out_tensor_dim[dims_lhs_ - 1];
    if (dims_lhs_ == 2 && dims_rhs_ == 3) {
      k = out_tensor_dim[1] * out_tensor_dim[2];
    }
    float alpha = 1.0f, beta = 0.0f;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;

    for (size_t i = 0; i < b; i++) {
      T* cur_in1 = in1 + i * m * n;
      T* cur_in2 = in2 + i * n * k;
      T* cur_out = out + i * m * k;
      // gradient respect to A
      HCTR_LIB_THROW(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k,
                                  &alpha, cur_in2, HIP_R_32F, k, cur_out, HIP_R_32F, k, &beta,
                                  cur_in1, HIP_R_32F, n, compute_type, HIPBLAS_GEMM_DEFAULT));

      cur_in1 = fprop_inputA_.get_ptr() + i * m * n;
      // gradient respect to B
      HCTR_LIB_THROW(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_T, k, n, m,
                                  &alpha, cur_out, HIP_R_32F, k, cur_in1, HIP_R_32F, n, &beta,
                                  cur_in2, HIP_R_32F, k, compute_type, HIPBLAS_GEMM_DEFAULT));
    }
  } else {
    T* in1 = input_tensors_[0].data<T>();
    T* in2 = input_tensors_[1].data<T>();
    T* out = output_tensors_[0].data<T>();

    const auto& input_tensor_shape = input_tensors_[0].shape();
    const auto& output_tensor_shape = output_tensors_[0].shape();

    int64_t m, n, k, b = 1;

    b = dims_lhs_ == 3 ? input_tensor_shape[0] : 1;
    b = dims_lhs_ == 4 ? input_tensor_shape[0] * input_tensor_shape[1] : b;
    m = input_tensor_shape[dims_lhs_ - 2];
    n = input_tensor_shape[dims_lhs_ - 1];
    k = output_tensor_shape[dims_lhs_ - 1];
    if (dims_lhs_ == 2 && dims_rhs_ == 3) {
      k = output_tensor_shape[1] * output_tensor_shape[2];
    }
    float alpha = 1.0f, beta = 0.0f;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;

    for (auto i = 0; i < b; i++) {
      T* cur_in1 = in1 + i * m * n;
      T* cur_in2 = in2 + i * n * k;
      T* cur_out = out + i * m * k;
      // gradient respect to A
      HCTR_LIB_THROW(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k,
                                  &alpha, cur_in2, HIP_R_32F, k, cur_out, HIP_R_32F, k, &beta,
                                  cur_in1, HIP_R_32F, n, compute_type, HIPBLAS_GEMM_DEFAULT));

      cur_in1 = fprop_inputA_tensor23_.data<T>() + i * m * n;
      // gradient respect to B
      HCTR_LIB_THROW(hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_T, k, n, m,
                                  &alpha, cur_out, HIP_R_32F, k, cur_in1, HIP_R_32F, n, &beta,
                                  cur_in2, HIP_R_32F, k, compute_type, HIPBLAS_GEMM_DEFAULT));
    }
  }
}

template class MatrixMultiplyLayer<float>;

}  // namespace HugeCTR
