#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common.hpp>
#include <cstdio>
#include <layers/fused_relu_bias_fully_connected_layer.hpp>
#include <linalg/reduce.cuh>
#include <utils.cuh>
#include <utils.hpp>

namespace HugeCTR {

namespace {

__global__ void reverse_relu_kernel(__half* dRelu, __half* mask, const __half* dY, size_t n) {
  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= n / 2) return;
  const size_t num_threads = blockDim.x * gridDim.x;
  const __half2 zero = TypeFunc<__half2>::zero();
  __half2* dRelu2 = reinterpret_cast<__half2*>(dRelu);
  __half2* mask2 = reinterpret_cast<__half2*>(mask);
  const __half2* dY2 = reinterpret_cast<const __half2*>(dY);
  __half2 m = __hgt2(mask2[tid], zero);
  dRelu2[tid] = __hmul2(__ldg(dY2 + tid), m);
  if (tid + num_threads >= n / 2) return;
  m = __hgt2(mask2[tid + num_threads], zero);
  dRelu2[tid + num_threads] = __hmul2(__ldg(dY2 + tid + num_threads), m);
}

__global__ void reverse_relu_kernel_not_aligned(__half* dRelu, __half* mask, const __half* dY,
                                                size_t n) {
  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= n) return;
  const __half zero = TypeFunc<__half>::zero();
  __half m = __hgt(mask[tid], zero);
  dRelu[tid] = __hmul(__ldg(dY + tid), m);
}

}  // namespace

FusedReluBiasFullyConnectedLayer::FusedReluBiasFullyConnectedLayer(
    const std::shared_ptr<BufferBlock2<float>>& master_weights_buff,
    const std::shared_ptr<BufferBlock2<__half>>& weights_buff,
    const std::shared_ptr<BufferBlock2<__half>>& weights_grad_buff,
    const std::shared_ptr<GeneralBuffer2<CudaAllocator>>& blobs_buff,
    const Tensor2<__half>& train_in_tensor, const Tensor2<__half>& mask_in_tensor,
    const Tensor2<__half>& dRelu_in_tensor, const Tensor2<__half>& db_in_tensor,
    const Tensor2<__half>& train_out_tensor, const Tensor2<__half>& mask_out_tensor,
    const Tensor2<__half>& dRelu_out_tensor, Tensor2<__half>& db_out_tensor,
    const std::shared_ptr<GPUResource>& gpu_resource, const FcPosition_t& pos,
    const Activation_t& act, const bool& skip_dgrad, std::vector<Initializer_t> initializer_types,
    const bool async_mlp_wgrad, const bool head_mask_in, const bool fuse_wb)
    : TrainableLayer<__half>(master_weights_buff, weights_buff, weights_grad_buff, gpu_resource,
                             initializer_types),
      balgo_k_(CUBLAS_GEMM_DEFAULT_TENSOR_OP),
      balgo_x_(CUBLAS_GEMM_DEFAULT_TENSOR_OP),
      balgo_b_(CUBLAS_GEMM_DEFAULT_TENSOR_OP),
      pos_(pos),
      act_(act),
      skip_dgrad_(skip_dgrad),
      async_mlp_wgrad_(async_mlp_wgrad),
      head_mask_in_(head_mask_in),
      fuse_wb_(fuse_wb),
      event_overlap_created_(false) {
  const auto& bottom_tensor_dim = train_in_tensor.get_dimensions();
  const auto& top_tensor_dim = train_out_tensor.get_dimensions();

  if (bottom_tensor_dim.size() != 2 || top_tensor_dim.size() != 2) {
    HCTR_OWN_THROW(Error_t::WrongInput, "input or output tensor doesn't has two dimensions");
  }

  size_t batch_size = bottom_tensor_dim[0];
  size_t output_size = top_tensor_dim[1];
  size_t input_size = bottom_tensor_dim[1];

  std::vector<size_t> kernel_dim = {input_size, output_size};
  std::vector<size_t> bias_dim = {1, output_size};
  std::vector<size_t> identity_dim = {1, batch_size};

  this->set_weight(0, kernel_dim);
  weights_half_.push_back(this->get_weight(0));
  this->set_weight(1, bias_dim);
  weights_half_.push_back(this->get_weight(1));
  this->set_wgrad(0, kernel_dim);
  weights_grad_.push_back(this->get_wgrad(0));
  this->set_wgrad(1, bias_dim);
  db_out_tensor = this->get_wgrad(1);
  weights_grad_.push_back(this->get_wgrad(1));

  blobs_buff->reserve(identity_dim, &identity_tensor_);

  train_in_tensor_ = train_in_tensor;
  //  if (pos_ == FcPosition_t::Head || pos_ == FcPosition_t::Isolated) {
  //    // mask_in_tensor_ = train_in_tensor;
  //  } else {
  mask_in_tensor_ = mask_in_tensor;
  dRelu_in_tensor_ = dRelu_in_tensor;
  db_in_tensor_ = db_in_tensor;
  //  }
  train_out_tensor_ = train_out_tensor;
  mask_out_tensor_ = mask_out_tensor;
  dRelu_out_tensor_ = dRelu_out_tensor;
  db_out_tensor_ = db_out_tensor;
  blobs_buff->reserve(kernel_dim, &bias_grad_tensor_);

  std::vector<size_t> mask_dim = {batch_size, output_size};
  blobs_buff->reserve(mask_dim, &mask_in_tensor_temp_);

  if (async_mlp_wgrad_)
    cublas_handle_wgrad_ = gpu_resource->get_cublas_handle_wgrad();
  else
    cublas_handle_wgrad_ = gpu_resource->get_cublas_handle();
}

void FusedReluBiasFullyConnectedLayer::initialize() {
  CudaDeviceContext context(get_device_id());
  HCTR_LIB_THROW(hipEventCreate(&event_overlap_));
  event_overlap_created_ = true;

  // TODO: We need different bottom desc based on is_train or not
  const auto& bottom_tensor_dim = get_bottom_tensor_fprop(true).get_dimensions();
  const auto& top_tensor_dim = train_out_tensor_.get_dimensions();
  __half* identity = identity_tensor_.get_ptr();

  int batch_size = bottom_tensor_dim[0];
  int output_size = top_tensor_dim[1];
  int input_size = bottom_tensor_dim[1];

  initialize_array<<<(batch_size - 1) / 1024 + 1, 1024, 0, get_gpu().get_stream()>>>(
      identity, batch_size, __float2half(1.0f));

  HCTR_LIB_THROW(hipblasLtMatmulDescCreate(&cublas_op_desc_, HIPBLAS_COMPUTE_32F, HIP_R_32F));

  hipblasOperation_t trans = HIPBLAS_OP_N;
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_, HIPBLASLT_MATMUL_DESC_TRANSA,
                                                &trans, sizeof(trans)));
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_, HIPBLASLT_MATMUL_DESC_TRANSB,
                                                &trans, sizeof(trans)));
  hipblasLtEpilogue_t epi = HIPBLASLT_EPILOGUE_RELU_AUX_BIAS;
  if (act_ == Activation_t::None) epi = HIPBLASLT_EPILOGUE_BIAS;
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_, HIPBLASLT_MATMUL_DESC_EPILOGUE,
                                                &epi, sizeof(epi)));
  const __half* bias = weights_half_[1].get_ptr();
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_, HIPBLASLT_MATMUL_DESC_BIAS_POINTER,
                                                &bias, sizeof(bias)));
  if (act_ != Activation_t::None) {
    __half* reluMask = mask_out_tensor_.get_ptr();
    hipblasLtMatmulDescSetAttribute(cublas_op_desc_, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER,
                                   &reluMask, sizeof(reluMask));
    long reluMaskLd = output_size;
    hipblasLtMatmulDescSetAttribute(cublas_op_desc_, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD,
                                   &reluMaskLd, sizeof(reluMaskLd));
  }

  HCTR_LIB_THROW(hipblasLtMatrixLayoutCreate(&cublas_kernel_desc_, HIP_R_16F, output_size,
                                            input_size, output_size));
  HCTR_LIB_THROW(hipblasLtMatrixLayoutCreate(&cublas_bottom_desc_, HIP_R_16F, input_size,
                                            batch_size, input_size));
  HCTR_LIB_THROW(hipblasLtMatrixLayoutCreate(&cublas_top_desc_, HIP_R_16F, output_size, batch_size,
                                            output_size));

  HCTR_LIB_THROW(hipblasLtMatmulPreferenceCreate(&cublas_preference_));

  cublaslt_workspace_size_ = 1024 * 1024 * 8;  // Set it to 8MB for now
  HCTR_LIB_THROW(hipMalloc(&cublaslt_workspace_, cublaslt_workspace_size_));
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(
      cublas_preference_, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &cublaslt_workspace_size_,
      sizeof(cublaslt_workspace_size_)));

  uint32_t pointer_mode = HIPBLASLT_POINTER_MODE_HOST;
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_, HIPBLASLT_MATMUL_DESC_POINTER_MODE,
                                                &pointer_mode, sizeof(pointer_mode)));

#if CUBLAS_VERSION < 120000
  pointer_mode = CUBLASLT_POINTER_MODE_MASK_HOST;
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(cublas_preference_,
                                                      CUBLASLT_MATMUL_PREF_POINTER_MODE_MASK,
                                                      &pointer_mode, sizeof(pointer_mode)));
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(
      cublas_preference_, CUBLASLT_MATMUL_PREF_EPILOGUE_MASK, &epi, sizeof(epi)));
#endif

  // By default set algo to best estimated heurstic
  hipblasLtMatmulHeuristicResult_t heuristic_result;
  int returned_res = 0;
  HCTR_LIB_THROW(hipblasLtMatmulAlgoGetHeuristic(
      get_gpu().get_cublaslt_handle(), cublas_op_desc_, cublas_kernel_desc_, cublas_bottom_desc_,
      cublas_top_desc_, cublas_top_desc_, cublas_preference_, 1, &heuristic_result, &returned_res));

  memcpy(&falgo_k_, &heuristic_result.algo, sizeof(falgo_k_));

  if (returned_res == 0) {
    HCTR_LIB_THROW(HIPBLAS_STATUS_NOT_SUPPORTED);
  }

  initialize_dgrad();
  initialize_wgrad();
}

void FusedReluBiasFullyConnectedLayer::initialize_dgrad() {
  // TODO: We need different bottom desc based on is_train or not
  const auto& bottom_tensor_dim = get_bottom_tensor_fprop(true).get_dimensions();
  const auto& top_tensor_dim = train_out_tensor_.get_dimensions();

  size_t batch_size = bottom_tensor_dim[0];
  size_t output_size = top_tensor_dim[1];
  size_t input_size = bottom_tensor_dim[1];

  HCTR_LIB_THROW(hipblasLtMatmulDescCreate(&cublas_op_desc_bprop_, HIPBLAS_COMPUTE_32F, HIP_R_32F));

  hipblasOperation_t transA = HIPBLAS_OP_T;
  hipblasOperation_t transB = HIPBLAS_OP_N;
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_bprop_, HIPBLASLT_MATMUL_DESC_TRANSA,
                                                &transA, sizeof(transA)));
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_bprop_, HIPBLASLT_MATMUL_DESC_TRANSB,
                                                &transB, sizeof(transB)));
  hipblasLtEpilogue_t epi;

  if (pos_ == FcPosition_t::Head || pos_ == FcPosition_t::Isolated) {
    epi = HIPBLASLT_EPILOGUE_DEFAULT;
    HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(
        cublas_op_desc_bprop_, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epi, sizeof(epi)));
  } else if (pos_ == FcPosition_t::Body || pos_ == FcPosition_t::Tail) {
    epi = fuse_wb_ ? CUBLASLT_EPILOGUE_DRELU : CUBLASLT_EPILOGUE_DRELU_BGRAD;
    hipblasLtMatmulDescSetAttribute(cublas_op_desc_bprop_, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epi,
                                   sizeof(epi));
    if (!fuse_wb_) {
      __half* bgrad = db_in_tensor_.get_ptr();
      hipblasLtMatmulDescSetAttribute(cublas_op_desc_bprop_, HIPBLASLT_MATMUL_DESC_BIAS_POINTER,
                                     &bgrad, sizeof(bgrad));
    }
    __half* reluMask = mask_in_tensor_.get_ptr();
    hipblasLtMatmulDescSetAttribute(cublas_op_desc_bprop_, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER,
                                   &reluMask, sizeof(reluMask));
    long reluMaskLd = input_size;
    hipblasLtMatmulDescSetAttribute(cublas_op_desc_bprop_, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD,
                                   &reluMaskLd, sizeof(reluMaskLd));
  }

  HCTR_LIB_THROW(hipblasLtMatrixLayoutCreate(&cublas_dRelu_top_desc_, HIP_R_16F, output_size,
                                            batch_size, output_size));
  HCTR_LIB_THROW(hipblasLtMatrixLayoutCreate(&cublas_dRelu_bottom_desc_, HIP_R_16F, input_size,
                                            batch_size, input_size));

  HCTR_LIB_THROW(hipblasLtMatmulPreferenceCreate(&cublas_preference_dRelu_));

  cublaslt_workspace_size_ = 1024 * 1024 * 8;  // Set it to 8MB for now
  HCTR_LIB_THROW(hipMalloc(&cublaslt_workspace_dRelu_, cublaslt_workspace_size_));
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(
      cublas_preference_dRelu_, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &cublaslt_workspace_size_,
      sizeof(cublaslt_workspace_size_)));

  uint32_t pointer_mode = HIPBLASLT_POINTER_MODE_HOST;
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_bprop_,
                                                HIPBLASLT_MATMUL_DESC_POINTER_MODE, &pointer_mode,
                                                sizeof(pointer_mode)));

#if CUBLAS_VERSION < 120000
  pointer_mode = CUBLASLT_POINTER_MODE_MASK_HOST;
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(cublas_preference_dRelu_,
                                                      CUBLASLT_MATMUL_PREF_POINTER_MODE_MASK,
                                                      &pointer_mode, sizeof(pointer_mode)));
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(
      cublas_preference_dRelu_, CUBLASLT_MATMUL_PREF_EPILOGUE_MASK, &epi, sizeof(epi)));
#endif

  // By default set algo to best estimated heurstic
  hipblasLtMatmulHeuristicResult_t heuristic_result;
  int returned_res = 0;
  HCTR_LIB_THROW(hipblasLtMatmulAlgoGetHeuristic(
      get_gpu().get_cublaslt_handle(), cublas_op_desc_bprop_, cublas_kernel_desc_,
      cublas_dRelu_top_desc_, cublas_dRelu_bottom_desc_, cublas_dRelu_bottom_desc_,
      cublas_preference_dRelu_, 1, &heuristic_result, &returned_res));

  memcpy(&balgo_dRelu_, &heuristic_result.algo, sizeof(balgo_dRelu_));

  if (returned_res == 0) {
    HCTR_LIB_THROW(HIPBLAS_STATUS_NOT_SUPPORTED);
  }
}

void FusedReluBiasFullyConnectedLayer::initialize_wgrad() {
  // TODO: We need different bottom desc based on is_train or not
  const auto& bottom_tensor_dim = get_bottom_tensor_fprop(true).get_dimensions();
  const auto& top_tensor_dim = train_out_tensor_.get_dimensions();
  size_t batch_size = bottom_tensor_dim[0];
  size_t output_size = top_tensor_dim[1];
  size_t input_size = bottom_tensor_dim[1];

  HCTR_LIB_THROW(hipblasLtMatmulDescCreate(&cublas_op_desc_wgrad_, HIPBLAS_COMPUTE_32F, HIP_R_32F));

  hipblasOperation_t transA = HIPBLAS_OP_N;
  hipblasOperation_t transB = HIPBLAS_OP_T;
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_wgrad_, HIPBLASLT_MATMUL_DESC_TRANSA,
                                                &transA, sizeof(transA)));
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_wgrad_, HIPBLASLT_MATMUL_DESC_TRANSB,
                                                &transB, sizeof(transB)));
  hipblasLtEpilogue_t epi;
  if (fuse_wb_ || pos_ == FcPosition_t::Tail || pos_ == FcPosition_t::Isolated) {
    epi = HIPBLASLT_EPILOGUE_BGRADA;
    __half* bgrad = db_out_tensor_.get_ptr();
    hipblasLtMatmulDescSetAttribute(cublas_op_desc_wgrad_, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bgrad,
                                   sizeof(bgrad));
  } else {
    epi = HIPBLASLT_EPILOGUE_DEFAULT;
  }

  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_wgrad_,
                                                HIPBLASLT_MATMUL_DESC_EPILOGUE, &epi, sizeof(epi)));

  HCTR_LIB_THROW(hipblasLtMatmulPreferenceCreate(&cublas_preference_wgrad_));

  cublaslt_workspace_size_ = 1024 * 1024 * 8;  // Set it to 8MB for now
  HCTR_LIB_THROW(hipMalloc(&cublaslt_workspace_wgrad_, cublaslt_workspace_size_));
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(
      cublas_preference_wgrad_, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &cublaslt_workspace_size_,
      sizeof(cublaslt_workspace_size_)));

  uint32_t pointer_mode = HIPBLASLT_POINTER_MODE_HOST;
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_wgrad_,
                                                HIPBLASLT_MATMUL_DESC_POINTER_MODE, &pointer_mode,
                                                sizeof(pointer_mode)));

#if CUBLAS_VERSION < 120000
  pointer_mode = CUBLASLT_POINTER_MODE_MASK_HOST;
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(cublas_preference_wgrad_,
                                                      CUBLASLT_MATMUL_PREF_POINTER_MODE_MASK,
                                                      &pointer_mode, sizeof(pointer_mode)));
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(
      cublas_preference_wgrad_, CUBLASLT_MATMUL_PREF_EPILOGUE_MASK, &epi, sizeof(epi)));
#endif

  // By default set algo to best estimated heurstic
  hipblasLtMatmulHeuristicResult_t heuristic_result;
  int returned_res = 0;
  HCTR_LIB_THROW(hipblasLtMatmulAlgoGetHeuristic(
      get_gpu().get_cublaslt_handle(), cublas_op_desc_wgrad_, cublas_dRelu_top_desc_,
      cublas_dRelu_bottom_desc_, cublas_kernel_desc_, cublas_kernel_desc_, cublas_preference_wgrad_,
      1, &heuristic_result, &returned_res));
  memcpy(&balgo_wgrad_, &heuristic_result.algo, sizeof(balgo_wgrad_));
  // returned_res is 0 indicates that there is no feasible algorithm.
  if (returned_res == 0) {
    HCTR_LIB_THROW(HIPBLAS_STATUS_NOT_SUPPORTED);
  }
}

void FusedReluBiasFullyConnectedLayer::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());

  const __half* kernel = weights_half_[0].get_ptr();
  const __half* bias = weights_half_[1].get_ptr();
  const __half* bottom = get_bottom_tensor_fprop(is_train).get_ptr();
  __half* top_fprop = train_out_tensor_.get_ptr();
  __half* mask_out = mask_out_tensor_.get_ptr();

  const auto& bottom_tensor_dim = get_bottom_tensor_fprop(is_train).get_dimensions();
  const auto& top_tensor_dim = train_out_tensor_.get_dimensions();

  size_t batch_size = bottom_tensor_dim[0];
  size_t output_size = top_tensor_dim[1];
  size_t input_size = bottom_tensor_dim[1];

  const float alpha = 1.0f;
  const float beta = 0.0f;

  HCTR_LIB_THROW(hipblasLtMatmul(
      get_gpu().get_cublaslt_handle(), cublas_op_desc_, &alpha, kernel, cublas_kernel_desc_, bottom,
      cublas_bottom_desc_, &beta, top_fprop, cublas_top_desc_, top_fprop, cublas_top_desc_,
      &falgo_k_, cublaslt_workspace_, cublaslt_workspace_size_, get_gpu().get_stream()));

  if ((pos_ == FcPosition_t::Tail || pos_ == FcPosition_t::Isolated) &&
      act_ != Activation_t::None) {
    size_t len = train_out_tensor_.get_num_elements();
    HCTR_LIB_THROW(hipMemcpyAsync(mask_out, top_fprop, len * sizeof(__half),
                                   hipMemcpyDeviceToDevice, get_gpu().get_stream()));
  }
}

void FusedReluBiasFullyConnectedLayer::bprop() {
  CudaDeviceContext context(get_device_id());

  const __half* kernel = weights_half_[0].get_ptr();
  const __half* train_out = train_out_tensor_.get_ptr();
  __half* mask_out = mask_out_tensor_.get_ptr();
  __half* kernel_grad = weights_grad_[0].get_ptr();
  __half* bias_grad = weights_grad_[1].get_ptr();
  __half* bottom = get_bottom_tensor_fprop(true).get_ptr();
  //__half* bottom_bprop = get_bottom_tensor_bprop(true).get_ptr();
  float* bias_grad_float = bias_grad_tensor_.get_ptr();
  __half* dRelu_top = dRelu_out_tensor_.get_ptr();
  const __half* identity = identity_tensor_.get_ptr();

  const auto& bottom_tensor_dim = get_bottom_tensor_fprop(true).get_dimensions();
  const auto& top_tensor_dim = train_out_tensor_.get_dimensions();

  size_t batch_size = bottom_tensor_dim[0];
  size_t output_size = top_tensor_dim[1];
  size_t input_size = bottom_tensor_dim[1];

  const float alpha = 1.0f;
  const float beta_k = 1.0f;
  const float beta_x = 0.0f;
  const float beta_b = 0.0f;

  // dRelu
  if (pos_ == FcPosition_t::Tail || pos_ == FcPosition_t::Isolated) {
    if (act_ != Activation_t::None) {
      if ((batch_size * output_size) % 4 == 0) {
        reverse_relu_kernel<<<(batch_size * output_size / 4 - 1) / 1024 + 1, 1024, 0,
                              get_gpu().get_stream()>>>(dRelu_top, mask_out, train_out,
                                                        batch_size * output_size);
      } else
        reverse_relu_kernel_not_aligned<<<(batch_size * output_size - 1) / 1024 + 1, 1024, 0,
                                          get_gpu().get_stream()>>>(dRelu_top, mask_out, train_out,
                                                                    batch_size * output_size);
    } else
      dRelu_top = train_out_tensor_.get_ptr();
  }

  // wait for dRelu
  if (async_mlp_wgrad_) {
    HCTR_LIB_THROW(hipEventRecord(event_overlap_, get_gpu().get_stream()));
    HCTR_LIB_THROW(hipStreamWaitEvent(get_gpu().get_comp_overlap_stream(), event_overlap_));
  }

  // bgrad+wgrad
  HCTR_LIB_THROW(hipblasLtMatmul(
      get_gpu().get_cublaslt_handle(), cublas_op_desc_wgrad_, &alpha, dRelu_top,
      cublas_dRelu_top_desc_, bottom, cublas_dRelu_bottom_desc_, &beta_k, kernel_grad,
      cublas_kernel_desc_, kernel_grad, cublas_kernel_desc_, &balgo_wgrad_,
      cublaslt_workspace_wgrad_, cublaslt_workspace_size_,
      async_mlp_wgrad_ ? get_gpu().get_comp_overlap_stream() : get_gpu().get_stream()));

  // dgrad
  if (!skip_dgrad_) {
    __half* bottom_bprop;
    if (head_mask_in_) {
      bottom_bprop = mask_in_tensor_.get_ptr();
    } else {
      bottom_bprop = train_in_tensor_.get_ptr();
    }

    if (pos_ == FcPosition_t::Body || pos_ == FcPosition_t::Tail) {
      bottom_bprop = dRelu_in_tensor_.get_ptr();
    }
    HCTR_LIB_THROW(hipblasLtMatmul(
        get_gpu().get_cublaslt_handle(), cublas_op_desc_bprop_, &alpha, kernel, cublas_kernel_desc_,
        dRelu_top, cublas_dRelu_top_desc_, &beta_x, bottom_bprop, cublas_dRelu_bottom_desc_,
        bottom_bprop, cublas_dRelu_bottom_desc_, &balgo_dRelu_, cublaslt_workspace_dRelu_,
        cublaslt_workspace_size_, get_gpu().get_stream()));
  }

  if (async_mlp_wgrad_ && pos_ == FcPosition_t::Head) {
    HCTR_LIB_THROW(hipEventRecord(event_overlap_, this->get_gpu().get_comp_overlap_stream()));
    HCTR_LIB_THROW(hipStreamWaitEvent(this->get_gpu().get_stream(), event_overlap_));
  }
}

void FusedReluBiasFullyConnectedLayer::search_algorithm() {
  // Set to the CUDA device where this layer assigned to
  CudaDeviceContext context(get_device_id());
  const size_t repeat_num = 100;
  const int max_algo_count = 16;

  // Device Tensors to be used
  __half* bottom = get_bottom_tensor_fprop(true).get_ptr();
  __half* top = train_out_tensor_.get_ptr();
  __half* kernel = weights_half_[0].get_ptr();
  __half* bias = weights_half_[1].get_ptr();
  __half* kernel_grad = weights_grad_[0].get_ptr();
  __half* bias_grad = weights_grad_[1].get_ptr();
  __half* identity = identity_tensor_.get_ptr();

  // Tensor dim
  const auto& bottom_tensor_dim = get_bottom_tensor_fprop(true).get_dimensions();
  const auto& top_tensor_dim = train_out_tensor_.get_dimensions();

  int batch_size = bottom_tensor_dim[0];
  int output_size = top_tensor_dim[1];
  int input_size = bottom_tensor_dim[1];

  // Record time for each algorithm
  float shortestTime = std::numeric_limits<float>::max();
  float time;
  hipEvent_t start, stop;
  HCTR_LIB_THROW(hipEventCreate(&start));
  HCTR_LIB_THROW(hipEventCreate(&stop));

  hipblasLtMatmulHeuristicResult_t heuristic_result[max_algo_count] = {0};
  int algo_count = 0;
  HCTR_LIB_THROW(hipblasLtMatmulAlgoGetHeuristic(
      get_gpu().get_cublaslt_handle(), cublas_op_desc_, cublas_kernel_desc_, cublas_bottom_desc_,
      cublas_top_desc_, cublas_top_desc_, cublas_preference_, max_algo_count, heuristic_result,
      &algo_count));

  if (algo_count == 0) {
    HCTR_LIB_THROW(HIPBLAS_STATUS_NOT_SUPPORTED);
  }

  for (int algoIdx = 0; algoIdx < algo_count; algoIdx++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 0.0f;
    HCTR_LIB_THROW(hipEventRecord(start, get_gpu().get_stream()));
    for (size_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status =
          hipblasLtMatmul(get_gpu().get_cublaslt_handle(), cublas_op_desc_, &alpha, kernel,
                         cublas_kernel_desc_, bottom, cublas_bottom_desc_, &beta, top,
                         cublas_top_desc_, top, cublas_top_desc_, &heuristic_result[algoIdx].algo,
                         cublaslt_workspace_, cublaslt_workspace_size_, get_gpu().get_stream());
    }
    HCTR_LIB_THROW(hipEventRecord(stop, get_gpu().get_stream()));
    HCTR_LIB_THROW(hipEventSynchronize(stop));
    HCTR_LIB_THROW(hipEventElapsedTime(&time, start, stop));

    // Avg Time(ms) for this algorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      //      HCTR_LOG(INFO, WORLD, "The algorithms %d is not supported for fprop, skipped.\n",
      //      testAlgo);
      continue;
    }

    // if(get_device_id()==0) HCTR_LOG(INFO, WORLD, "Algo: %d, wavesCount: %f, time: %f\n",
    //           (int)heuristic_result[algoIdx].algo,
    //           heuristic_result[algoIdx].wavesCount,
    //           time);
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      memcpy(&falgo_k_, &heuristic_result[algoIdx].algo, sizeof(falgo_k_));
      // if(get_device_id()==0) HCTR_LOG(INFO, WORLD, "Picked algorithm: %d",
      // heuristic_result[algoIdx].algo);
    }
  }

  // dRelu in backward pass
  // Reset shortestTime
  shortestTime = std::numeric_limits<float>::max();
  hipblasLtMatmulHeuristicResult_t heuristic_result_dRelu[max_algo_count] = {0};
  int algo_count_dRelu = 0;
  HCTR_LIB_THROW(hipblasLtMatmulAlgoGetHeuristic(
      get_gpu().get_cublaslt_handle(), cublas_op_desc_bprop_, cublas_kernel_desc_,
      cublas_dRelu_top_desc_, cublas_dRelu_bottom_desc_, cublas_dRelu_bottom_desc_,
      cublas_preference_dRelu_, max_algo_count, heuristic_result_dRelu, &algo_count_dRelu));

  if (algo_count_dRelu == 0) {
    HCTR_LIB_THROW(HIPBLAS_STATUS_NOT_SUPPORTED);
  }

  for (int algoIdx = 0; algoIdx < algo_count_dRelu; algoIdx++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 0.0f;
    HCTR_LIB_THROW(hipEventRecord(start, get_gpu().get_stream()));
    for (size_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasLtMatmul(get_gpu().get_cublaslt_handle(), cublas_op_desc_bprop_, &alpha,
                              kernel, cublas_kernel_desc_, top, cublas_dRelu_top_desc_, &beta,
                              bottom, cublas_dRelu_bottom_desc_, bottom, cublas_dRelu_bottom_desc_,
                              &heuristic_result_dRelu[algoIdx].algo, cublaslt_workspace_dRelu_,
                              cublaslt_workspace_size_, get_gpu().get_stream());
    }
    HCTR_LIB_THROW(hipEventRecord(stop, get_gpu().get_stream()));
    HCTR_LIB_THROW(hipEventSynchronize(stop));
    HCTR_LIB_THROW(hipEventElapsedTime(&time, start, stop));

    // Avg Time(ms) for this algorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      //      HCTR_LOG(INFO, WORLD, "The algorithms %d is not supported for fprop, skipped.\n",
      //      testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      memcpy(&balgo_dRelu_, &heuristic_result_dRelu[algoIdx].algo, sizeof(balgo_dRelu_));
    }
  }

  // wgrad in backward pass
  // Reset shortestTime
  shortestTime = std::numeric_limits<float>::max();
  hipblasLtMatmulHeuristicResult_t heuristic_result_wgrad[max_algo_count] = {0};
  int algo_count_wgrad = 0;
  HCTR_LIB_THROW(hipblasLtMatmulAlgoGetHeuristic(
      get_gpu().get_cublaslt_handle(), cublas_op_desc_wgrad_, cublas_dRelu_top_desc_,
      cublas_dRelu_bottom_desc_, cublas_kernel_desc_, cublas_kernel_desc_, cublas_preference_wgrad_,
      max_algo_count, heuristic_result_wgrad, &algo_count_wgrad));

  if (algo_count_wgrad == 0) {
    HCTR_LIB_THROW(HIPBLAS_STATUS_NOT_SUPPORTED);
  }

  for (int algoIdx = 0; algoIdx < algo_count_wgrad; algoIdx++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 1.0f;
    HCTR_LIB_THROW(hipEventRecord(start, get_gpu().get_stream()));
    for (size_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasLtMatmul(get_gpu().get_cublaslt_handle(), cublas_op_desc_wgrad_, &alpha, top,
                              cublas_dRelu_top_desc_, bottom, cublas_dRelu_bottom_desc_, &beta,
                              kernel, cublas_kernel_desc_, kernel, cublas_kernel_desc_,
                              &heuristic_result_wgrad[algoIdx].algo, cublaslt_workspace_wgrad_,
                              cublaslt_workspace_size_, get_gpu().get_stream());
    }
    HCTR_LIB_THROW(hipEventRecord(stop, get_gpu().get_stream()));
    HCTR_LIB_THROW(hipEventSynchronize(stop));
    HCTR_LIB_THROW(hipEventElapsedTime(&time, start, stop));

    // Avg Time(ms) for this algorithm for fprop GEMM
    time = time / repeat_num;
    // HCTR_LOG(INFO, WORLD, "algoIdx: %d, time: %f, shortest time: %f\n", algoIdx, time,
    // shortestTime); Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      //      HCTR_LOG(INFO, WORLD, "The algorithms %d is not supported for fprop, skipped.\n",
      //      testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      // HCTR_LOG(INFO, WORLD, "wgrad cublasMatmul algoIdx: %d, time: %f\n", algoIdx, shortestTime);
      memcpy(&balgo_wgrad_, &heuristic_result_wgrad[algoIdx].algo, sizeof(balgo_wgrad_));
    }
  }

  // Reset shortestTime
  shortestTime = std::numeric_limits<float>::max();

  // Start, end for search
  const hipblasGemmAlgo_t startAlgo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
  const hipblasGemmAlgo_t endAlgo = CUBLAS_GEMM_ALGO15_TENSOR_OP;

  // Search all the algorithm for balgo_k_
  for (int testAlgo = startAlgo; testAlgo <= endAlgo; testAlgo++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 1.0f;

    // Record start event
    HCTR_LIB_THROW(hipEventRecord(start, get_gpu().get_stream()));
    for (size_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_T, output_size,
                            input_size, batch_size, &alpha, top, HIP_R_16F, output_size, bottom,
                            HIP_R_16F, input_size, &beta, kernel_grad, HIP_R_16F, output_size,
                            HIP_R_32F, static_cast<hipblasGemmAlgo_t>(testAlgo));
    }
    HCTR_LIB_THROW(hipEventRecord(stop, get_gpu().get_stream()));
    HCTR_LIB_THROW(hipEventSynchronize(stop));
    HCTR_LIB_THROW(hipEventElapsedTime(&time, start, stop));
    // Avg Time(ms) for this algorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      //      HCTR_LOG(INFO, WORLD, "The algorithms %d is not supported for bprop_W, skipped.\n",
      //      testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      // HCTR_LOG(INFO, WORLD, "wgrad hipblasGemmEx algoIdx: %d, time: %f\n", testAlgo,
      // shortestTime);
      balgo_k_ = static_cast<hipblasGemmAlgo_t>(testAlgo);
    }
  }

  // Reset shortestTime
  shortestTime = std::numeric_limits<float>::max();

  // Search all the algorithm for balgo_b_
  for (int testAlgo = startAlgo; testAlgo <= endAlgo; testAlgo++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Record start event
    HCTR_LIB_THROW(hipEventRecord(start, get_gpu().get_stream()));
    for (size_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N, output_size, 1,
                            batch_size, &alpha, top, HIP_R_16F, output_size, identity, HIP_R_16F,
                            batch_size, &beta, bias_grad, HIP_R_16F, output_size, HIP_R_32F,
                            static_cast<hipblasGemmAlgo_t>(testAlgo));
    }
    HCTR_LIB_THROW(hipEventRecord(stop, get_gpu().get_stream()));
    HCTR_LIB_THROW(hipEventSynchronize(stop));
    HCTR_LIB_THROW(hipEventElapsedTime(&time, start, stop));
    // Avg Time(ms) for this algorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      // HCTR_LOG(INFO, WORLD, "The algorithms %d is not supported for bprop_W, skipped.\n",
      // testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      balgo_b_ = static_cast<hipblasGemmAlgo_t>(testAlgo);
    }
  }
  // Reset shortestTime
  shortestTime = std::numeric_limits<float>::max();

  // Search all the algorithm for balgo_x_
  for (int testAlgo = startAlgo; testAlgo <= endAlgo; testAlgo++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const __half alpha = 1.0f;
    const __half beta = 0.0f;

    // Record start event
    HCTR_LIB_THROW(hipEventRecord(start, get_gpu().get_stream()));
    for (size_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N, input_size,
                            batch_size, output_size, &alpha, kernel, HIP_R_16F, output_size, top,
                            HIP_R_16F, output_size, &beta, bottom, HIP_R_16F, input_size,
                            HIP_R_32F, static_cast<hipblasGemmAlgo_t>(testAlgo));
    }

    HCTR_LIB_THROW(hipEventRecord(stop, get_gpu().get_stream()));
    HCTR_LIB_THROW(hipEventSynchronize(stop));
    HCTR_LIB_THROW(hipEventElapsedTime(&time, start, stop));
    // Avg Time(ms) for this algorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      //      HCTR_LOG(INFO, WORLD, "The algorithms %d is not supported for bprop_Xn, skipped.\n",
      //      testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      balgo_x_ = static_cast<hipblasGemmAlgo_t>(testAlgo);
    }
  }

  // Print selection information
  // HCTR_LOG(INFO, WORLD, "The algorithm selection for falgo_k_, balgo_k_, balgo_x_ are: %d, %d and
  // %d.\n",
  //        (int)falgo_k_ - CUBLAS_GEMM_DEFAULT_TENSOR_OP,
  //        (int)balgo_k_ - CUBLAS_GEMM_DEFAULT_TENSOR_OP,
  //        (int)balgo_x_ - CUBLAS_GEMM_DEFAULT_TENSOR_OP);

  // Output msg
  // HCTR_LOG(INFO, ROOT, "The fully-connected layer has finished choosing the algorithm for cublas
  // Gemm.\n"); Clean-up
  HCTR_LIB_THROW(hipEventDestroy(start));
  HCTR_LIB_THROW(hipEventDestroy(stop));
}  // namespace HugeCTR

std::unique_ptr<DataSimulator> FusedReluBiasFullyConnectedLayer::get_uniform_initializer(
    const int index) {
  size_t bottom_dim = get_bottom_tensor_fprop(true).get_dimensions()[1];
  size_t top_dim = train_out_tensor_.get_dimensions()[1];

  float limit = 1.0f / ((0 == index ? bottom_dim : 0) + top_dim);
  return std::make_unique<UniformDataSimulator>(-1 * limit, limit);
}

std::unique_ptr<DataSimulator> FusedReluBiasFullyConnectedLayer::get_xavier_uniform_initializer(
    const int index) {
  size_t bottom_dim = get_bottom_tensor_fprop(true).get_dimensions()[1];
  size_t top_dim = train_out_tensor_.get_dimensions()[1];

  return std::make_unique<VarianceScalingSimulator>(1.f, data_simu::Mode_t::Fan_avg,
                                                    data_simu::Distribution_t::Uniform,
                                                    0 == index ? bottom_dim : 0, top_dim);
}

std::unique_ptr<DataSimulator> FusedReluBiasFullyConnectedLayer::get_xavier_norm_initializer(
    const int index) {
  size_t bottom_dim = get_bottom_tensor_fprop(true).get_dimensions()[1];
  size_t top_dim = train_out_tensor_.get_dimensions()[1];

  return std::make_unique<VarianceScalingSimulator>(1.f, data_simu::Mode_t::Fan_avg,
                                                    data_simu::Distribution_t::Norm,
                                                    0 == index ? bottom_dim : 0, top_dim);
}

std::unique_ptr<DataSimulator> FusedReluBiasFullyConnectedLayer::get_default_initializer(
    const int index) {
  size_t bottom_dim = get_bottom_tensor_fprop(true).get_dimensions()[1];
  size_t top_dim = train_out_tensor_.get_dimensions()[1];

  std::unique_ptr<DataSimulator> simu(nullptr);
  if (0 == index) {
    simu.reset(new VarianceScalingSimulator(1.f, data_simu::Mode_t::Fan_avg,
                                            data_simu::Distribution_t::Norm, bottom_dim, top_dim));
  } else if (1 == index) {
    float stddev = sqrt(1.f / top_dim);
    simu.reset(new GaussianDataSimulator(0, stddev, -2 * stddev, 2 * stddev));
  } else {
    HCTR_OWN_THROW(Error_t::OutOfBound, "index != {0, 1}.");
  }

  return simu;
}

Core23TempFusedReluBiasFullyConnectedLayer::Core23TempFusedReluBiasFullyConnectedLayer(
    const core23::Tensor& train_in_tensor, const core23::Tensor& mask_in_tensor,
    const core23::Tensor& dRelu_in_tensor, const core23::Tensor& db_in_tensor,
    const core23::Tensor& train_out_tensor, const core23::Tensor& mask_out_tensor,
    const core23::Tensor& dRelu_out_tensor, core23::Tensor& db_out_tensor,
    const std::shared_ptr<GPUResource>& gpu_resource, const FcPosition_t& pos,
    const Activation_t& act, const bool& skip_dgrad, std::vector<Initializer_t> initializer_types,
    const bool async_mlp_wgrad, const bool head_mask_in, const bool fuse_wb)
    : Core23TempTrainableLayer<__half>({train_in_tensor}, {train_out_tensor}, gpu_resource,
                                       initializer_types),
      balgo_k_(CUBLAS_GEMM_DEFAULT_TENSOR_OP),
      balgo_x_(CUBLAS_GEMM_DEFAULT_TENSOR_OP),
      balgo_b_(CUBLAS_GEMM_DEFAULT_TENSOR_OP),
      pos_(pos),
      act_(act),
      skip_dgrad_(skip_dgrad),
      async_mlp_wgrad_(async_mlp_wgrad),
      head_mask_in_(head_mask_in),
      fuse_wb_(fuse_wb),
      event_overlap_created_(false) {
  const auto& bottom_tensor_dim = train_in_tensor.shape();
  const auto& top_tensor_dim = train_out_tensor.shape();

  if (bottom_tensor_dim.dims() != 2 || top_tensor_dim.dims() != 2) {
    HCTR_OWN_THROW(Error_t::WrongInput, "input or output tensor doesn't has two dimensions");
  }

  int64_t batch_size = bottom_tensor_dim.size(0);
  int64_t output_size = top_tensor_dim.size(1);
  int64_t input_size = bottom_tensor_dim.size(1);

  core23::Shape kernel_dim = {input_size, output_size};
  core23::Shape bias_dim = {1, output_size};
  core23::Shape identity_dim = {1, batch_size};

  this->set_weight(0, kernel_dim);
  weights_half_.push_back(this->get_weight(0));
  this->set_weight(1, bias_dim);
  weights_half_.push_back(this->get_weight(1));
  this->set_wgrad(0, kernel_dim);
  weights_grad_.push_back(this->get_wgrad(0));
  this->set_wgrad(1, bias_dim);
  db_out_tensor = this->get_wgrad(1);
  weights_grad_.push_back(this->get_wgrad(1));

  core23::BufferParams blobs_buffer_params = {};
  blobs_buffer_params.channel = GetBlobsBufferChannel();
  core23::Device device(core23::DeviceType::GPU, gpu_resource->get_device_id());

  identity_tensor_ = core23::Tensor(core23::TensorParams()
                                        .data_type(core23::ToScalarType<__half>::value)
                                        .shape(identity_dim)
                                        .device(device)
                                        .buffer_params(blobs_buffer_params));

  //  if (pos_ == FcPosition_t::Head || pos_ == FcPosition_t::Isolated) {
  //    // mask_in_tensor_ = train_in_tensor;
  //  } else {
  mask_in_tensor_ = mask_in_tensor;
  dRelu_in_tensor_ = dRelu_in_tensor;
  db_in_tensor_ = db_in_tensor;
  //  }
  mask_out_tensor_ = mask_out_tensor;
  dRelu_out_tensor_ = dRelu_out_tensor;
  db_out_tensor_ = db_out_tensor;

  if (async_mlp_wgrad_)
    cublas_handle_wgrad_ = gpu_resource->get_cublas_handle_wgrad();
  else
    cublas_handle_wgrad_ = gpu_resource->get_cublas_handle();
}

void Core23TempFusedReluBiasFullyConnectedLayer::initialize() {
  CudaDeviceContext context(get_device_id());
  HCTR_LIB_THROW(hipEventCreate(&event_overlap_));
  event_overlap_created_ = true;

  // TODO: We need different bottom desc based on is_train or not
  const auto& bottom_tensor_dim = get_bottom_tensor_fprop(true).shape();
  const auto& top_tensor_dim = this->output_tensors_[0].shape();
  __half* identity = identity_tensor_.data<__half>();

  int batch_size = bottom_tensor_dim.size(0);
  int output_size = top_tensor_dim.size(1);
  int input_size = bottom_tensor_dim.size(1);

  initialize_array<<<(batch_size - 1) / 1024 + 1, 1024, 0, get_gpu().get_stream()>>>(
      identity, batch_size, __float2half(1.0f));

  HCTR_LIB_THROW(hipblasLtMatmulDescCreate(&cublas_op_desc_, HIPBLAS_COMPUTE_32F, HIP_R_32F));

  hipblasOperation_t trans = HIPBLAS_OP_N;
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_, HIPBLASLT_MATMUL_DESC_TRANSA,
                                                &trans, sizeof(trans)));
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_, HIPBLASLT_MATMUL_DESC_TRANSB,
                                                &trans, sizeof(trans)));
  hipblasLtEpilogue_t epi = HIPBLASLT_EPILOGUE_RELU_AUX_BIAS;
  if (act_ == Activation_t::None) epi = HIPBLASLT_EPILOGUE_BIAS;
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_, HIPBLASLT_MATMUL_DESC_EPILOGUE,
                                                &epi, sizeof(epi)));
  const __half* bias = weights_half_[1].data<__half>();
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_, HIPBLASLT_MATMUL_DESC_BIAS_POINTER,
                                                &bias, sizeof(bias)));
  if (act_ != Activation_t::None) {
    __half* reluMask = mask_out_tensor_.data<__half>();
    hipblasLtMatmulDescSetAttribute(cublas_op_desc_, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER,
                                   &reluMask, sizeof(reluMask));
    long reluMaskLd = output_size;
    hipblasLtMatmulDescSetAttribute(cublas_op_desc_, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD,
                                   &reluMaskLd, sizeof(reluMaskLd));
  }

  HCTR_LIB_THROW(hipblasLtMatrixLayoutCreate(&cublas_kernel_desc_, HIP_R_16F, output_size,
                                            input_size, output_size));
  HCTR_LIB_THROW(hipblasLtMatrixLayoutCreate(&cublas_bottom_desc_, HIP_R_16F, input_size,
                                            batch_size, input_size));
  HCTR_LIB_THROW(hipblasLtMatrixLayoutCreate(&cublas_top_desc_, HIP_R_16F, output_size, batch_size,
                                            output_size));

  HCTR_LIB_THROW(hipblasLtMatmulPreferenceCreate(&cublas_preference_));

  cublaslt_workspace_size_ = 1024 * 1024 * 8;  // Set it to 8MB for now
  HCTR_LIB_THROW(hipMalloc(&cublaslt_workspace_, cublaslt_workspace_size_));
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(
      cublas_preference_, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &cublaslt_workspace_size_,
      sizeof(cublaslt_workspace_size_)));

  uint32_t pointer_mode = HIPBLASLT_POINTER_MODE_HOST;
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_, HIPBLASLT_MATMUL_DESC_POINTER_MODE,
                                                &pointer_mode, sizeof(pointer_mode)));

#if CUBLAS_VERSION < 120000
  pointer_mode = CUBLASLT_POINTER_MODE_MASK_HOST;
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(cublas_preference_,
                                                      CUBLASLT_MATMUL_PREF_POINTER_MODE_MASK,
                                                      &pointer_mode, sizeof(pointer_mode)));
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(
      cublas_preference_, CUBLASLT_MATMUL_PREF_EPILOGUE_MASK, &epi, sizeof(epi)));
#endif

  // By default set algo to best estimated heurstic
  hipblasLtMatmulHeuristicResult_t heuristic_result;
  int returned_res = 0;
  HCTR_LIB_THROW(hipblasLtMatmulAlgoGetHeuristic(
      get_gpu().get_cublaslt_handle(), cublas_op_desc_, cublas_kernel_desc_, cublas_bottom_desc_,
      cublas_top_desc_, cublas_top_desc_, cublas_preference_, 1, &heuristic_result, &returned_res));

  memcpy(&falgo_k_, &heuristic_result.algo, sizeof(falgo_k_));

  if (returned_res == 0) {
    HCTR_LIB_THROW(HIPBLAS_STATUS_NOT_SUPPORTED);
  }

  initialize_dgrad();
  initialize_wgrad();
}

void Core23TempFusedReluBiasFullyConnectedLayer::initialize_dgrad() {
  // TODO: We need different bottom desc based on is_train or not
  const auto& bottom_tensor_dim = get_bottom_tensor_fprop(true).shape();
  const auto& top_tensor_dim = this->output_tensors_[0].shape();

  int64_t batch_size = bottom_tensor_dim.size(0);
  int64_t output_size = top_tensor_dim.size(1);
  int64_t input_size = bottom_tensor_dim.size(1);

  HCTR_LIB_THROW(hipblasLtMatmulDescCreate(&cublas_op_desc_bprop_, HIPBLAS_COMPUTE_32F, HIP_R_32F));

  hipblasOperation_t transA = HIPBLAS_OP_T;
  hipblasOperation_t transB = HIPBLAS_OP_N;
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_bprop_, HIPBLASLT_MATMUL_DESC_TRANSA,
                                                &transA, sizeof(transA)));
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_bprop_, HIPBLASLT_MATMUL_DESC_TRANSB,
                                                &transB, sizeof(transB)));
  hipblasLtEpilogue_t epi;

  if (pos_ == FcPosition_t::Head || pos_ == FcPosition_t::Isolated) {
    epi = HIPBLASLT_EPILOGUE_DEFAULT;
    HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(
        cublas_op_desc_bprop_, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epi, sizeof(epi)));
  } else if (pos_ == FcPosition_t::Body || pos_ == FcPosition_t::Tail) {
    epi = fuse_wb_ ? CUBLASLT_EPILOGUE_DRELU : CUBLASLT_EPILOGUE_DRELU_BGRAD;
    hipblasLtMatmulDescSetAttribute(cublas_op_desc_bprop_, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epi,
                                   sizeof(epi));
    if (!fuse_wb_) {
      __half* bgrad = db_in_tensor_.data<__half>();
      hipblasLtMatmulDescSetAttribute(cublas_op_desc_bprop_, HIPBLASLT_MATMUL_DESC_BIAS_POINTER,
                                     &bgrad, sizeof(bgrad));
    }
    __half* reluMask = mask_in_tensor_.data<__half>();
    hipblasLtMatmulDescSetAttribute(cublas_op_desc_bprop_, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER,
                                   &reluMask, sizeof(reluMask));
    long reluMaskLd = input_size;
    hipblasLtMatmulDescSetAttribute(cublas_op_desc_bprop_, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD,
                                   &reluMaskLd, sizeof(reluMaskLd));
  }

  HCTR_LIB_THROW(hipblasLtMatrixLayoutCreate(&cublas_dRelu_top_desc_, HIP_R_16F, output_size,
                                            batch_size, output_size));
  HCTR_LIB_THROW(hipblasLtMatrixLayoutCreate(&cublas_dRelu_bottom_desc_, HIP_R_16F, input_size,
                                            batch_size, input_size));

  HCTR_LIB_THROW(hipblasLtMatmulPreferenceCreate(&cublas_preference_dRelu_));

  cublaslt_workspace_size_ = 1024 * 1024 * 8;  // Set it to 8MB for now
  HCTR_LIB_THROW(hipMalloc(&cublaslt_workspace_dRelu_, cublaslt_workspace_size_));
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(
      cublas_preference_dRelu_, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &cublaslt_workspace_size_,
      sizeof(cublaslt_workspace_size_)));

  uint32_t pointer_mode = HIPBLASLT_POINTER_MODE_HOST;
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_bprop_,
                                                HIPBLASLT_MATMUL_DESC_POINTER_MODE, &pointer_mode,
                                                sizeof(pointer_mode)));

#if CUBLAS_VERSION < 120000
  pointer_mode = CUBLASLT_POINTER_MODE_MASK_HOST;
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(cublas_preference_dRelu_,
                                                      CUBLASLT_MATMUL_PREF_POINTER_MODE_MASK,
                                                      &pointer_mode, sizeof(pointer_mode)));
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(
      cublas_preference_dRelu_, CUBLASLT_MATMUL_PREF_EPILOGUE_MASK, &epi, sizeof(epi)));
#endif

  // By default set algo to best estimated heurstic
  hipblasLtMatmulHeuristicResult_t heuristic_result;
  int returned_res = 0;
  HCTR_LIB_THROW(hipblasLtMatmulAlgoGetHeuristic(
      get_gpu().get_cublaslt_handle(), cublas_op_desc_bprop_, cublas_kernel_desc_,
      cublas_dRelu_top_desc_, cublas_dRelu_bottom_desc_, cublas_dRelu_bottom_desc_,
      cublas_preference_dRelu_, 1, &heuristic_result, &returned_res));

  memcpy(&balgo_dRelu_, &heuristic_result.algo, sizeof(balgo_dRelu_));

  if (returned_res == 0) {
    HCTR_LIB_THROW(HIPBLAS_STATUS_NOT_SUPPORTED);
  }
}

void Core23TempFusedReluBiasFullyConnectedLayer::initialize_wgrad() {
  // TODO: We need different bottom desc based on is_train or not
  const auto& bottom_tensor_dim = get_bottom_tensor_fprop(true).shape();
  const auto& top_tensor_dim = this->output_tensors_[0].shape();
  int64_t batch_size = bottom_tensor_dim.size(0);
  int64_t output_size = top_tensor_dim.size(1);
  int64_t input_size = bottom_tensor_dim.size(1);

  HCTR_LIB_THROW(hipblasLtMatmulDescCreate(&cublas_op_desc_wgrad_, HIPBLAS_COMPUTE_32F, HIP_R_32F));

  hipblasOperation_t transA = HIPBLAS_OP_N;
  hipblasOperation_t transB = HIPBLAS_OP_T;
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_wgrad_, HIPBLASLT_MATMUL_DESC_TRANSA,
                                                &transA, sizeof(transA)));
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_wgrad_, HIPBLASLT_MATMUL_DESC_TRANSB,
                                                &transB, sizeof(transB)));
  hipblasLtEpilogue_t epi;
  if (fuse_wb_ || pos_ == FcPosition_t::Tail || pos_ == FcPosition_t::Isolated) {
    epi = HIPBLASLT_EPILOGUE_BGRADA;
    __half* bgrad = db_out_tensor_.data<__half>();
    hipblasLtMatmulDescSetAttribute(cublas_op_desc_wgrad_, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bgrad,
                                   sizeof(bgrad));
  } else {
    epi = HIPBLASLT_EPILOGUE_DEFAULT;
  }

  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_wgrad_,
                                                HIPBLASLT_MATMUL_DESC_EPILOGUE, &epi, sizeof(epi)));

  HCTR_LIB_THROW(hipblasLtMatmulPreferenceCreate(&cublas_preference_wgrad_));

  cublaslt_workspace_size_ = 1024 * 1024 * 8;  // Set it to 8MB for now
  HCTR_LIB_THROW(hipMalloc(&cublaslt_workspace_wgrad_, cublaslt_workspace_size_));
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(
      cublas_preference_wgrad_, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &cublaslt_workspace_size_,
      sizeof(cublaslt_workspace_size_)));

  uint32_t pointer_mode = HIPBLASLT_POINTER_MODE_HOST;
  HCTR_LIB_THROW(hipblasLtMatmulDescSetAttribute(cublas_op_desc_wgrad_,
                                                HIPBLASLT_MATMUL_DESC_POINTER_MODE, &pointer_mode,
                                                sizeof(pointer_mode)));

#if CUBLAS_VERSION < 120000
  pointer_mode = CUBLASLT_POINTER_MODE_MASK_HOST;
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(cublas_preference_wgrad_,
                                                      CUBLASLT_MATMUL_PREF_POINTER_MODE_MASK,
                                                      &pointer_mode, sizeof(pointer_mode)));
  HCTR_LIB_THROW(hipblasLtMatmulPreferenceSetAttribute(
      cublas_preference_wgrad_, CUBLASLT_MATMUL_PREF_EPILOGUE_MASK, &epi, sizeof(epi)));
#endif

  // By default set algo to best estimated heurstic
  hipblasLtMatmulHeuristicResult_t heuristic_result;
  int returned_res = 0;
  HCTR_LIB_THROW(hipblasLtMatmulAlgoGetHeuristic(
      get_gpu().get_cublaslt_handle(), cublas_op_desc_wgrad_, cublas_dRelu_top_desc_,
      cublas_dRelu_bottom_desc_, cublas_kernel_desc_, cublas_kernel_desc_, cublas_preference_wgrad_,
      1, &heuristic_result, &returned_res));
  memcpy(&balgo_wgrad_, &heuristic_result.algo, sizeof(balgo_wgrad_));
  // returned_res is 0 indicates that there is no feasible algorithm.
  if (returned_res == 0) {
    HCTR_LIB_THROW(HIPBLAS_STATUS_NOT_SUPPORTED);
  }
}

void Core23TempFusedReluBiasFullyConnectedLayer::fprop(bool is_train) {
  CudaDeviceContext context(get_device_id());

  const __half* kernel = weights_half_[0].data<__half>();
  const __half* bias = weights_half_[1].data<__half>();
  const __half* bottom = get_bottom_tensor_fprop(is_train).data<__half>();
  __half* top_fprop = this->output_tensors_[0].data<__half>();
  __half* mask_out = mask_out_tensor_.data<__half>();

  const auto& bottom_tensor_dim = get_bottom_tensor_fprop(is_train).shape();
  const auto& top_tensor_dim = this->output_tensors_[0].shape();

  int64_t batch_size = bottom_tensor_dim.size(0);
  int64_t output_size = top_tensor_dim.size(1);
  int64_t input_size = bottom_tensor_dim.size(1);

  const float alpha = 1.0f;
  const float beta = 0.0f;

  HCTR_LIB_THROW(hipblasLtMatmul(
      get_gpu().get_cublaslt_handle(), cublas_op_desc_, &alpha, kernel, cublas_kernel_desc_, bottom,
      cublas_bottom_desc_, &beta, top_fprop, cublas_top_desc_, top_fprop, cublas_top_desc_,
      &falgo_k_, cublaslt_workspace_, cublaslt_workspace_size_, get_gpu().get_stream()));

  if ((pos_ == FcPosition_t::Tail || pos_ == FcPosition_t::Isolated) &&
      act_ != Activation_t::None) {
    int64_t len = this->output_tensors_[0].num_elements();
    HCTR_LIB_THROW(hipMemcpyAsync(mask_out, top_fprop, len * sizeof(__half),
                                   hipMemcpyDeviceToDevice, get_gpu().get_stream()));
  }
}

void Core23TempFusedReluBiasFullyConnectedLayer::bprop() {
  CudaDeviceContext context(get_device_id());

  const __half* kernel = weights_half_[0].data<__half>();
  const __half* train_out = this->output_tensors_[0].data<__half>();
  __half* mask_out = mask_out_tensor_.data<__half>();
  __half* kernel_grad = weights_grad_[0].data<__half>();
  __half* bias_grad = weights_grad_[1].data<__half>();
  __half* bottom = get_bottom_tensor_fprop(true).data<__half>();
  //__half* bottom_bprop = get_bottom_tensor_bprop(true).data<__half>();

  __half* dRelu_top = dRelu_out_tensor_.data<__half>();
  const __half* identity = identity_tensor_.data<__half>();

  const auto& bottom_tensor_dim = get_bottom_tensor_fprop(true).shape();
  const auto& top_tensor_dim = this->output_tensors_[0].shape();

  int64_t batch_size = bottom_tensor_dim.size(0);
  int64_t output_size = top_tensor_dim.size(1);
  int64_t input_size = bottom_tensor_dim.size(1);

  const float alpha = 1.0f;
  const float beta_k = 1.0f;
  const float beta_x = 0.0f;
  const float beta_b = 0.0f;

  // dRelu
  if (pos_ == FcPosition_t::Tail || pos_ == FcPosition_t::Isolated) {
    if (act_ != Activation_t::None) {
      if ((batch_size * output_size) % 4 == 0) {
        reverse_relu_kernel<<<(batch_size * output_size / 4 - 1) / 1024 + 1, 1024, 0,
                              get_gpu().get_stream()>>>(dRelu_top, mask_out, train_out,
                                                        batch_size * output_size);
      } else
        reverse_relu_kernel_not_aligned<<<(batch_size * output_size - 1) / 1024 + 1, 1024, 0,
                                          get_gpu().get_stream()>>>(dRelu_top, mask_out, train_out,
                                                                    batch_size * output_size);
    } else
      dRelu_top = this->output_tensors_[0].data<__half>();
  }

  // wait for dRelu
  if (async_mlp_wgrad_) {
    HCTR_LIB_THROW(hipEventRecord(event_overlap_, get_gpu().get_stream()));
    HCTR_LIB_THROW(hipStreamWaitEvent(get_gpu().get_comp_overlap_stream(), event_overlap_));
  }

  // bgrad+wgrad
  HCTR_LIB_THROW(hipblasLtMatmul(
      get_gpu().get_cublaslt_handle(), cublas_op_desc_wgrad_, &alpha, dRelu_top,
      cublas_dRelu_top_desc_, bottom, cublas_dRelu_bottom_desc_, &beta_k, kernel_grad,
      cublas_kernel_desc_, kernel_grad, cublas_kernel_desc_, &balgo_wgrad_,
      cublaslt_workspace_wgrad_, cublaslt_workspace_size_,
      async_mlp_wgrad_ ? get_gpu().get_comp_overlap_stream() : get_gpu().get_stream()));

  // dgrad
  if (!skip_dgrad_) {
    __half* bottom_bprop;
    if (head_mask_in_) {
      bottom_bprop = mask_in_tensor_.data<__half>();
    } else {
      bottom_bprop = this->input_tensors_[0].data<__half>();
    }

    if (pos_ == FcPosition_t::Body || pos_ == FcPosition_t::Tail) {
      bottom_bprop = dRelu_in_tensor_.data<__half>();
    }
    HCTR_LIB_THROW(hipblasLtMatmul(
        get_gpu().get_cublaslt_handle(), cublas_op_desc_bprop_, &alpha, kernel, cublas_kernel_desc_,
        dRelu_top, cublas_dRelu_top_desc_, &beta_x, bottom_bprop, cublas_dRelu_bottom_desc_,
        bottom_bprop, cublas_dRelu_bottom_desc_, &balgo_dRelu_, cublaslt_workspace_dRelu_,
        cublaslt_workspace_size_, get_gpu().get_stream()));
  }

  if (async_mlp_wgrad_ && pos_ == FcPosition_t::Head) {
    HCTR_LIB_THROW(hipEventRecord(event_overlap_, this->get_gpu().get_comp_overlap_stream()));
    HCTR_LIB_THROW(hipStreamWaitEvent(this->get_gpu().get_stream(), event_overlap_));
  }
}

void Core23TempFusedReluBiasFullyConnectedLayer::search_algorithm() {
  // Set to the CUDA device where this layer assigned to
  CudaDeviceContext context(get_device_id());
  const int64_t repeat_num = 100;
  const int max_algo_count = 16;

  // Device Tensors to be used
  __half* bottom = get_bottom_tensor_fprop(true).data<__half>();
  __half* top = this->output_tensors_[0].data<__half>();
  __half* kernel = weights_half_[0].data<__half>();
  __half* bias = weights_half_[1].data<__half>();
  __half* kernel_grad = weights_grad_[0].data<__half>();
  __half* bias_grad = weights_grad_[1].data<__half>();
  __half* identity = identity_tensor_.data<__half>();

  // Tensor dim
  const auto& bottom_tensor_dim = get_bottom_tensor_fprop(true).shape();
  const auto& top_tensor_dim = this->output_tensors_[0].shape();

  int batch_size = bottom_tensor_dim.size(0);
  int output_size = top_tensor_dim.size(1);
  int input_size = bottom_tensor_dim.size(1);

  // Record time for each algorithm
  float shortestTime = std::numeric_limits<float>::max();
  float time;
  hipEvent_t start, stop;
  HCTR_LIB_THROW(hipEventCreate(&start));
  HCTR_LIB_THROW(hipEventCreate(&stop));

  hipblasLtMatmulHeuristicResult_t heuristic_result[max_algo_count] = {0};
  int algo_count = 0;
  HCTR_LIB_THROW(hipblasLtMatmulAlgoGetHeuristic(
      get_gpu().get_cublaslt_handle(), cublas_op_desc_, cublas_kernel_desc_, cublas_bottom_desc_,
      cublas_top_desc_, cublas_top_desc_, cublas_preference_, max_algo_count, heuristic_result,
      &algo_count));

  if (algo_count == 0) {
    HCTR_LIB_THROW(HIPBLAS_STATUS_NOT_SUPPORTED);
  }

  for (int algoIdx = 0; algoIdx < algo_count; algoIdx++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 0.0f;
    HCTR_LIB_THROW(hipEventRecord(start, get_gpu().get_stream()));
    for (int64_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status =
          hipblasLtMatmul(get_gpu().get_cublaslt_handle(), cublas_op_desc_, &alpha, kernel,
                         cublas_kernel_desc_, bottom, cublas_bottom_desc_, &beta, top,
                         cublas_top_desc_, top, cublas_top_desc_, &heuristic_result[algoIdx].algo,
                         cublaslt_workspace_, cublaslt_workspace_size_, get_gpu().get_stream());
    }
    HCTR_LIB_THROW(hipEventRecord(stop, get_gpu().get_stream()));
    HCTR_LIB_THROW(hipEventSynchronize(stop));
    HCTR_LIB_THROW(hipEventElapsedTime(&time, start, stop));

    // Avg Time(ms) for this algorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      //      HCTR_LOG(INFO, WORLD, "The algorithms %d is not supported for fprop, skipped.\n",
      //      testAlgo);
      continue;
    }

    // if(get_device_id()==0) HCTR_LOG(INFO, WORLD, "Algo: %d, wavesCount: %f, time: %f\n",
    //           (int)heuristic_result[algoIdx].algo,
    //           heuristic_result[algoIdx].wavesCount,
    //           time);
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      memcpy(&falgo_k_, &heuristic_result[algoIdx].algo, sizeof(falgo_k_));
      // if(get_device_id()==0) HCTR_LOG(INFO, WORLD, "Picked algorithm: %d",
      // heuristic_result[algoIdx].algo);
    }
  }

  // dRelu in backward pass
  // Reset shortestTime
  shortestTime = std::numeric_limits<float>::max();
  hipblasLtMatmulHeuristicResult_t heuristic_result_dRelu[max_algo_count] = {0};
  int algo_count_dRelu = 0;
  HCTR_LIB_THROW(hipblasLtMatmulAlgoGetHeuristic(
      get_gpu().get_cublaslt_handle(), cublas_op_desc_bprop_, cublas_kernel_desc_,
      cublas_dRelu_top_desc_, cublas_dRelu_bottom_desc_, cublas_dRelu_bottom_desc_,
      cublas_preference_dRelu_, max_algo_count, heuristic_result_dRelu, &algo_count_dRelu));

  if (algo_count_dRelu == 0) {
    HCTR_LIB_THROW(HIPBLAS_STATUS_NOT_SUPPORTED);
  }

  for (int algoIdx = 0; algoIdx < algo_count_dRelu; algoIdx++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 0.0f;
    HCTR_LIB_THROW(hipEventRecord(start, get_gpu().get_stream()));
    for (int64_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasLtMatmul(get_gpu().get_cublaslt_handle(), cublas_op_desc_bprop_, &alpha,
                              kernel, cublas_kernel_desc_, top, cublas_dRelu_top_desc_, &beta,
                              bottom, cublas_dRelu_bottom_desc_, bottom, cublas_dRelu_bottom_desc_,
                              &heuristic_result_dRelu[algoIdx].algo, cublaslt_workspace_dRelu_,
                              cublaslt_workspace_size_, get_gpu().get_stream());
    }
    HCTR_LIB_THROW(hipEventRecord(stop, get_gpu().get_stream()));
    HCTR_LIB_THROW(hipEventSynchronize(stop));
    HCTR_LIB_THROW(hipEventElapsedTime(&time, start, stop));

    // Avg Time(ms) for this algorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      //      HCTR_LOG(INFO, WORLD, "The algorithms %d is not supported for fprop, skipped.\n",
      //      testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      memcpy(&balgo_dRelu_, &heuristic_result_dRelu[algoIdx].algo, sizeof(balgo_dRelu_));
    }
  }

  // wgrad in backward pass
  // Reset shortestTime
  shortestTime = std::numeric_limits<float>::max();
  hipblasLtMatmulHeuristicResult_t heuristic_result_wgrad[max_algo_count] = {0};
  int algo_count_wgrad = 0;
  HCTR_LIB_THROW(hipblasLtMatmulAlgoGetHeuristic(
      get_gpu().get_cublaslt_handle(), cublas_op_desc_wgrad_, cublas_dRelu_top_desc_,
      cublas_dRelu_bottom_desc_, cublas_kernel_desc_, cublas_kernel_desc_, cublas_preference_wgrad_,
      max_algo_count, heuristic_result_wgrad, &algo_count_wgrad));

  if (algo_count_wgrad == 0) {
    HCTR_LIB_THROW(HIPBLAS_STATUS_NOT_SUPPORTED);
  }

  for (int algoIdx = 0; algoIdx < algo_count_wgrad; algoIdx++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 1.0f;
    HCTR_LIB_THROW(hipEventRecord(start, get_gpu().get_stream()));
    for (int64_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasLtMatmul(get_gpu().get_cublaslt_handle(), cublas_op_desc_wgrad_, &alpha, top,
                              cublas_dRelu_top_desc_, bottom, cublas_dRelu_bottom_desc_, &beta,
                              kernel, cublas_kernel_desc_, kernel, cublas_kernel_desc_,
                              &heuristic_result_wgrad[algoIdx].algo, cublaslt_workspace_wgrad_,
                              cublaslt_workspace_size_, get_gpu().get_stream());
    }
    HCTR_LIB_THROW(hipEventRecord(stop, get_gpu().get_stream()));
    HCTR_LIB_THROW(hipEventSynchronize(stop));
    HCTR_LIB_THROW(hipEventElapsedTime(&time, start, stop));

    // Avg Time(ms) for this algorithm for fprop GEMM
    time = time / repeat_num;
    // HCTR_LOG(INFO, WORLD, "algoIdx: %d, time: %f, shortest time: %f\n", algoIdx, time,
    // shortestTime); Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      //      HCTR_LOG(INFO, WORLD, "The algorithms %d is not supported for fprop, skipped.\n",
      //      testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      // HCTR_LOG(INFO, WORLD, "wgrad cublasMatmul algoIdx: %d, time: %f\n", algoIdx, shortestTime);
      memcpy(&balgo_wgrad_, &heuristic_result_wgrad[algoIdx].algo, sizeof(balgo_wgrad_));
    }
  }

  // Reset shortestTime
  shortestTime = std::numeric_limits<float>::max();

  // Start, end for search
  const hipblasGemmAlgo_t startAlgo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
  const hipblasGemmAlgo_t endAlgo = CUBLAS_GEMM_ALGO15_TENSOR_OP;

  // Search all the algorithm for balgo_k_
  for (int testAlgo = startAlgo; testAlgo <= endAlgo; testAlgo++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 1.0f;

    // Record start event
    HCTR_LIB_THROW(hipEventRecord(start, get_gpu().get_stream()));
    for (int64_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_T, output_size,
                            input_size, batch_size, &alpha, top, HIP_R_16F, output_size, bottom,
                            HIP_R_16F, input_size, &beta, kernel_grad, HIP_R_16F, output_size,
                            HIP_R_32F, static_cast<hipblasGemmAlgo_t>(testAlgo));
    }
    HCTR_LIB_THROW(hipEventRecord(stop, get_gpu().get_stream()));
    HCTR_LIB_THROW(hipEventSynchronize(stop));
    HCTR_LIB_THROW(hipEventElapsedTime(&time, start, stop));
    // Avg Time(ms) for this algorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      //      HCTR_LOG(INFO, WORLD, "The algorithms %d is not supported for bprop_W, skipped.\n",
      //      testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      // HCTR_LOG(INFO, WORLD, "wgrad hipblasGemmEx algoIdx: %d, time: %f\n", testAlgo,
      // shortestTime);
      balgo_k_ = static_cast<hipblasGemmAlgo_t>(testAlgo);
    }
  }

  // Reset shortestTime
  shortestTime = std::numeric_limits<float>::max();

  // Search all the algorithm for balgo_b_
  for (int testAlgo = startAlgo; testAlgo <= endAlgo; testAlgo++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Record start event
    HCTR_LIB_THROW(hipEventRecord(start, get_gpu().get_stream()));
    for (int64_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N, output_size, 1,
                            batch_size, &alpha, top, HIP_R_16F, output_size, identity, HIP_R_16F,
                            batch_size, &beta, bias_grad, HIP_R_16F, output_size, HIP_R_32F,
                            static_cast<hipblasGemmAlgo_t>(testAlgo));
    }
    HCTR_LIB_THROW(hipEventRecord(stop, get_gpu().get_stream()));
    HCTR_LIB_THROW(hipEventSynchronize(stop));
    HCTR_LIB_THROW(hipEventElapsedTime(&time, start, stop));
    // Avg Time(ms) for this algorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      // HCTR_LOG(INFO, WORLD, "The algorithms %d is not supported for bprop_W, skipped.\n",
      // testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      balgo_b_ = static_cast<hipblasGemmAlgo_t>(testAlgo);
    }
  }
  // Reset shortestTime
  shortestTime = std::numeric_limits<float>::max();

  // Search all the algorithm for balgo_x_
  for (int testAlgo = startAlgo; testAlgo <= endAlgo; testAlgo++) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    const __half alpha = 1.0f;
    const __half beta = 0.0f;

    // Record start event
    HCTR_LIB_THROW(hipEventRecord(start, get_gpu().get_stream()));
    for (int64_t i = 0; i < repeat_num && status == HIPBLAS_STATUS_SUCCESS; ++i) {
      status = hipblasGemmEx(get_gpu().get_cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N, input_size,
                            batch_size, output_size, &alpha, kernel, HIP_R_16F, output_size, top,
                            HIP_R_16F, output_size, &beta, bottom, HIP_R_16F, input_size,
                            HIP_R_32F, static_cast<hipblasGemmAlgo_t>(testAlgo));
    }

    HCTR_LIB_THROW(hipEventRecord(stop, get_gpu().get_stream()));
    HCTR_LIB_THROW(hipEventSynchronize(stop));
    HCTR_LIB_THROW(hipEventElapsedTime(&time, start, stop));
    // Avg Time(ms) for this algorithm for fprop GEMM
    time = time / repeat_num;
    // Skip if the algorithm is supported for fprop configuration
    if (status != HIPBLAS_STATUS_SUCCESS) {
      //      HCTR_LOG(INFO, WORLD, "The algorithms %d is not supported for bprop_Xn, skipped.\n",
      //      testAlgo);
      continue;
    }
    // Record the optimal time and algorithm
    if (time < shortestTime) {
      shortestTime = time;
      balgo_x_ = static_cast<hipblasGemmAlgo_t>(testAlgo);
    }
  }

  // Print selection information
  // HCTR_LOG(INFO, WORLD, "The algorithm selection for falgo_k_, balgo_k_, balgo_x_ are: %d, %d and
  // %d.\n",
  //        (int)falgo_k_ - CUBLAS_GEMM_DEFAULT_TENSOR_OP,
  //        (int)balgo_k_ - CUBLAS_GEMM_DEFAULT_TENSOR_OP,
  //        (int)balgo_x_ - CUBLAS_GEMM_DEFAULT_TENSOR_OP);

  // Output msg
  // HCTR_LOG(INFO, ROOT, "The fully-connected layer has finished choosing the algorithm for cublas
  // Gemm.\n"); Clean-up
  HCTR_LIB_THROW(hipEventDestroy(start));
  HCTR_LIB_THROW(hipEventDestroy(stop));
}  // namespace HugeCTR

std::unique_ptr<DataSimulator> Core23TempFusedReluBiasFullyConnectedLayer::get_uniform_initializer(
    const int index) {
  int64_t bottom_dim = get_bottom_tensor_fprop(true).shape().size(1);
  int64_t top_dim = this->output_tensors_[0].shape().size(1);

  float limit = 1.0f / ((0 == index ? bottom_dim : 0) + top_dim);
  return std::make_unique<UniformDataSimulator>(-1 * limit, limit);
}

std::unique_ptr<DataSimulator>
Core23TempFusedReluBiasFullyConnectedLayer::get_xavier_uniform_initializer(const int index) {
  int64_t bottom_dim = get_bottom_tensor_fprop(true).shape().size(1);
  int64_t top_dim = this->output_tensors_[0].shape().size(1);

  return std::make_unique<VarianceScalingSimulator>(1.f, data_simu::Mode_t::Fan_avg,
                                                    data_simu::Distribution_t::Uniform,
                                                    0 == index ? bottom_dim : 0, top_dim);
}

std::unique_ptr<DataSimulator>
Core23TempFusedReluBiasFullyConnectedLayer::get_xavier_norm_initializer(const int index) {
  int64_t bottom_dim = get_bottom_tensor_fprop(true).shape().size(1);
  int64_t top_dim = this->output_tensors_[0].shape().size(1);

  return std::make_unique<VarianceScalingSimulator>(1.f, data_simu::Mode_t::Fan_avg,
                                                    data_simu::Distribution_t::Norm,
                                                    0 == index ? bottom_dim : 0, top_dim);
}

std::unique_ptr<DataSimulator> Core23TempFusedReluBiasFullyConnectedLayer::get_default_initializer(
    const int index) {
  int64_t bottom_dim = get_bottom_tensor_fprop(true).shape().size(1);
  int64_t top_dim = this->output_tensors_[0].shape().size(1);

  std::unique_ptr<DataSimulator> simu(nullptr);
  if (0 == index) {
    simu.reset(new VarianceScalingSimulator(1.f, data_simu::Mode_t::Fan_avg,
                                            data_simu::Distribution_t::Norm, bottom_dim, top_dim));
  } else if (1 == index) {
    float stddev = sqrt(1.f / top_dim);
    simu.reset(new GaussianDataSimulator(0, stddev, -2 * stddev, 2 * stddev));
  } else {
    HCTR_OWN_THROW(Error_t::OutOfBound, "index != {0, 1}.");
  }

  return simu;
}

}  // namespace HugeCTR
