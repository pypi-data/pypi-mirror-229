#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <core23/tensor.hpp>
#include <regularizer.hpp>
#include <utility>

namespace HugeCTR {

template <typename T>
Regularizer<T>::Regularizer(const Tensor2<float>& weight_buff, const Tensor2<T>& wgrad_buff,
                            const int batch_size, const std::shared_ptr<GPUResource>& gpu_resource)
    : weight_buff_(weight_buff),
      wgrad_buff_(wgrad_buff),
      weight_tensors_(std::nullopt),
      wgrad_tensors_(std::nullopt),
      batch_size_(batch_size),
      gpu_resource_(gpu_resource) {}

template <typename T>
Regularizer<T>::Regularizer(std::optional<WeightTensors> weight_tensors,
                            std::optional<WgradTensors<T>> wgrad_tensors, const int batch_size,
                            const std::shared_ptr<GPUResource>& gpu_resource)
    : weight_tensors_(weight_tensors),
      wgrad_tensors_(wgrad_tensors),
      batch_size_(batch_size),
      gpu_resource_(gpu_resource) {}

template <typename T>
void Regularizer<T>::compute_rterm() {
  CudaDeviceContext context(get_device_id());
  if (weight_tensors_) {
    // core23 branch
    auto flat_weight_tensor = weight_tensors_->flatten();
    const float* weight = flat_weight_tensor.data();
    auto num_elements = flat_weight_tensor.size(0);
    do_compute_rterm(weight, &h_rterm_, num_elements);
    return;
  } else if (weight_buff_.allocated()) {
    // legacy branch
    const float* weight = weight_buff_.get_ptr();
    auto num_elements = weight_buff_.get_num_elements();
    do_compute_rterm(weight, &h_rterm_, num_elements);
    return;
  } else {
    do_compute_rterm(nullptr, &h_rterm_, 0);
  }
}

template <typename T>
void Regularizer<T>::initialize_wgrad() {
  CudaDeviceContext context(get_device_id());
  // no regularizer
  if (!weight_tensors_ && !weight_buff_.allocated()) {
    return;
  }
  if (!wgrad_tensors_) {
    const float* weight = weight_buff_.get_ptr();
    T* wgrad = wgrad_buff_.get_ptr();
    auto num_elements = weight_buff_.get_num_elements();
    do_initialize_wgrad(weight, wgrad, num_elements, get_gpu().get_stream());
  } else {
    auto flat_weight_tensor = weight_tensors_->flatten();
    auto flat_wgrad_tensor = wgrad_tensors_->flatten();
    const float* weight = flat_weight_tensor.data();
    T* wgrad = flat_wgrad_tensor.data();
    auto num_elements = flat_weight_tensor.size(0);
    do_initialize_wgrad(weight, wgrad, num_elements, get_gpu().get_stream());
  }
}

template class Regularizer<float>;
template class Regularizer<__half>;
}  // namespace HugeCTR
